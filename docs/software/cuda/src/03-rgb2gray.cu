#include "hip/hip_runtime.h"
//
// CUDA program to convert RGB image to grayscale.
//
//
#include "rgb2gray.hpp"
#include <Magick++.h> 
#include <iostream>   
using namespace std;
using namespace Magick;

//
// TODO #1
// Add the CUDA Kernel Device code to convert the RGB image to grayscale.
//
//
__global__ void convert(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, unsigned char *d_gray, int width, int height)
{

}

__host__ std::tuple<unsigned char *, unsigned char *, unsigned char *, unsigned char *> allocateDeviceMemory(int rows, int columns)
{
    cout << "Allocating GPU device memory\n";
    int num_image_pixels = rows * columns;
    size_t size = num_image_pixels * sizeof(unsigned char);

    // Allocate the device input vector d_r
    unsigned char *d_r = NULL;
    hipError_t err = hipMalloc(&d_r, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_r (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector d_g
    unsigned char *d_g = NULL;
    err = hipMalloc(&d_g, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_g (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector d_b
    unsigned char *d_b = NULL;
    err = hipMalloc(&d_b, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector d_gray
    unsigned char *d_gray = NULL;
    err = hipMalloc(&d_gray, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_gray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate device constant symbols for rows and columns
    hipMemcpyToSymbol(HIP_SYMBOL(d_rows), &rows, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_columns), &columns, sizeof(int), 0, hipMemcpyHostToDevice);

    return {d_r, d_g, d_b, d_gray};
}


__host__ void copyFromHostToDevice(unsigned char *h_r, unsigned char *d_r, unsigned char *h_g, unsigned char *d_g, unsigned char *h_b, unsigned char *d_b, int rows, int columns)
{
    cout << "Copying from Host to Device\n";
    int num_image_pixels = rows * columns;
    size_t size = num_image_pixels * sizeof(unsigned char);

    hipError_t err;
    err = hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector r from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_g, h_g, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector g from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector b from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void executeKernel(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, unsigned char *d_gray, int rows, int columns, int threadsPerBlock)
{
    cout << "Executing kernel\n";

    // Calculate grid and block dimensions
    dim3 block(threadsPerBlock, threadsPerBlock, 1);
    dim3 grid(ceil((float) columns / block.x), ceil((float) rows / block.y), 1);

    cout << "threadsPerBlock = " << threadsPerBlock << " x " << threadsPerBlock << endl;
    cout << "grid = " << grid.x << " " << grid.y << ", " << "block = " << block.x << " " << block.y << endl;
    cout << "total pixels = " << rows * columns << ", total threads = " << grid.x * grid.y * block.x * block.y << endl;
    
    // 
    // TODO #2 
    // Call the CUDA kernel that converts the RGB image to gray scale.
    //

    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch convert kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void copyFromDeviceToHost(unsigned char *d_gray, unsigned char *gray, int rows, int columns)
{
    cout << "Copying from Device to Host\n";
    // Copy the device result int array in device memory to the host result int array in host memory.
    size_t size = rows * columns * sizeof(unsigned char);

    hipError_t err = hipMemcpy(gray, d_gray, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy array d_gray from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Free device global memory
__host__ void deallocateMemory(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, unsigned char *d_gray)
{
    cout << "Deallocating GPU device memory\n";
    hipError_t err = hipFree(d_r);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_r (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_g);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_g (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_b);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_gray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device int variable d_image_num_pixels (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    cout << "Cleaning CUDA device\n";
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ std::tuple<std::string, std::string, int> parseCommandLineArguments(int argc, char *argv[])
{
    cout << "Parsing CLI arguments\n";
    int threadsPerBlock = 16;
    std::string inputImage = "images/rainbow_2048.png";
    std::string outputImage = "grey.png";

    for (int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if (option.compare("-i") == 0)
        {
            inputImage = value;
        }
        else if (option.compare("-o") == 0)
        {
            outputImage = value;
        }
        else if (option.compare("-t") == 0)
        {
            threadsPerBlock = atoi(value.c_str());
        }
    }

    cout << "inputImage: " << inputImage << " outputImage: " << outputImage << " threadsPerBlock dimension: " << threadsPerBlock << "\n";
    return {inputImage, outputImage, threadsPerBlock};
}

__host__ std::tuple<int, int, unsigned char *, unsigned char *, unsigned char *> readImageFromFile(std::string inputFile)
{
    cout << "Reading Image From File\n";
    Image img;
    img.read(inputFile);

    const int rows = img.rows();
    const int columns = img.columns();
    const int channels = img.depth();

    cout << "Rows: " << rows << " Columns: " << columns << " Channels: " << channels << "\n";

    unsigned char *h_r = (unsigned char *)malloc(sizeof(unsigned char) * rows * columns);
    unsigned char *h_g = (unsigned char *)malloc(sizeof(unsigned char) * rows * columns);
    unsigned char *h_b = (unsigned char *)malloc(sizeof(unsigned char) * rows * columns);
    cout << "malloc passed" << endl;
    
    for(int r = 0; r < rows; ++r)
    {
        for(int c = 0; c < columns; ++c)
        {
            Color pixel = img.pixelColor(c, r);
            h_r[r*rows+c] = (int) 255 * (float) pixel.redQuantum() / QuantumRange;
            h_g[r*rows+c] = (int) 255 * (float) pixel.greenQuantum() / QuantumRange;
            h_b[r*rows+c] = (int) 255 * (float) pixel.blueQuantum() / QuantumRange;
        }
    }

    cout << "Finished reading image into RGB arrays\n";

    return {rows, columns, h_r, h_g, h_b};
}

int main(int argc, char *argv[])
{
    std::tuple<std::string, std::string, int> parsedCommandLineArgsTuple = parseCommandLineArguments(argc, argv);
    std::string inputImage = get<0>(parsedCommandLineArgsTuple);
    std::string outputImage = get<1>(parsedCommandLineArgsTuple);
    int threadsPerBlock = get<2>(parsedCommandLineArgsTuple);
    try 
    {
        auto[rows, columns, h_r, h_g, h_b] = readImageFromFile(inputImage);
        cout << "Finshed reading image file." << endl;

        unsigned char *gray = (unsigned char *)malloc(sizeof(unsigned char) * rows * columns);
        std::tuple<unsigned char *, unsigned char *, unsigned char *, unsigned char *> memoryTuple = allocateDeviceMemory(rows, columns);
        unsigned char *d_r = get<0>(memoryTuple);
        unsigned char *d_g = get<1>(memoryTuple);
        unsigned char *d_b = get<2>(memoryTuple);
        unsigned char *d_gray = get<3>(memoryTuple);

        copyFromHostToDevice(h_r, d_r, h_g, d_g, h_b, d_b, rows, columns);

        executeKernel(d_r, d_g, d_b, d_gray, rows, columns, threadsPerBlock);

        copyFromDeviceToHost(d_gray, gray, rows, columns);
        deallocateMemory(d_r, d_g, d_b, d_gray);
        cleanUpDevice();

        InitializeMagick(*argv);
        Image image;
        // Each pixel is GRAY, 8 bytes (4 unsigned shorts per pixel).
        vector<unsigned short> rawPixels(columns * rows * 4); 

        for (size_t i = 0; i < columns * rows; ++i) {
            rawPixels[i * 4 + 0] = gray[i] * 257; // Red
            rawPixels[i * 4 + 1] = gray[i] * 257; // Green
            rawPixels[i * 4 + 2] = gray[i] * 257; // Blue
            rawPixels[i * 4 + 3] = 65535; // Alpha (fully opaque)
        }
        
        // Create an Image from the gray data.
        Blob my_blob(rawPixels.data(), rawPixels.size() * sizeof(unsigned short));
        image.size(Geometry(columns, rows));
	    // Specify the pixel format (Red, Green, Blue, Alpha)
        image.magick("RGBA"); 
        image.read(my_blob);
	    // Convert the image from rgba to png.
	    image.magick("PNG"); 

        cout << "Writing output image to: " << outputImage << endl;
        image.write(outputImage);
    }
    catch (Exception &error_)
    {
        cout << "Caught exception: " << error_.what() << endl;
        return 1;
    }
    return 0;
}
